#define N 256
#include <stdio.h>

__global void vecAdd (int *a, int *b, int *c);

int main() {
	int a[N], b[N], c[N];
	int *dev_A, *dev_b, *dev_c;

	for (int i = 0; i < 240; i++) {
		a[i] = i;
		b[i] = i**2;
	}
	size = N * sizeof(int);

	hipMalloc((void**)&dev_a, size);
	hipMalloc((void**)&dev_b, size);
	hipMalloc((void**)&dev_c, size);

	cudeMemcpy (dev_a, a, size, hipMemcpyHostToDevice);
	cudeMemcpy (dev_b, b, size, hipMemcpyHostToDevice);

	vectAdd<<<1,N>>>(dev_a,dev_b,dev_c);

	hipMemcpy (c, dev_c, size,hipMemcpyDeviceToHost);

	cudeFree(dev_a);
	cudeFree(dev_b);
	hipFree(dev_c);

	exit(0);
}

__global void vecAdd (int *a, int *b, int *c) {
	int i = threadIdx.x;
	c[i] = a[i] + b[i];
}
