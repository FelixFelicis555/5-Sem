#define N 256

#include <hip/hip_runtime.h>
#include <stdio.h>
__global__ void vecAdd (int *a, int *b, int *c);

int main() {
	int a[N], b[N], c[N];
	int *dev_a, *dev_b, *dev_c;
	// initialize a and b with real values (NOT SHOWN)
	int size = N * sizeof(int);
	for (int i = 0; i < 200; i++) {
		a[i] = i;
		b[i] = i/2;
	}
	hipMalloc((void**)&dev_a, size);
	hipMalloc((void**)&dev_b, size);
	hipMalloc((void**)&dev_c, size);

	hipMemcpy(dev_a, a, size,hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, size,hipMemcpyHostToDevice);
	vecAdd<<<1,N>>>(dev_a,dev_b,dev_c);
	hipMemcpy(c, dev_c, size,hipMemcpyDeviceToHost);

	for (int i = 0; i < 200; i++) {
		printf("%d + %d = %d\n", a[i], b[i], c[i]);
	}
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
	exit (0);
}

__global__ void vecAdd (int *a, int *b, int *c) {
	int i = threadIdx.x;
	c[i] = a[i] + b[i];
}
