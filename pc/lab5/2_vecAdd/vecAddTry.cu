#include "hip/hip_runtime.h"
#define N 2048
#include <stdio.h>

__global__ void vecAdd (int *a, int *b, int *c);
void printArray(int a[], int b[], int c[]);

int main() {
	int a[N], b[N], c[N];
	int *dev_a, *dev_b, *dev_c;

	// initialize a and b with real values (NOT SHOWN)
	int size = N * sizeof(int);
	for (int i = 0; i < N; i++) {
		a[i] = i;
		b[i] = i/2;
	}
	for(int T=128;T<=1024;T+=128){
		clock_t t_exec = clock();
		hipMalloc((void**)&dev_a, size);
		hipMalloc((void**)&dev_b, size);
		hipMalloc((void**)&dev_c, size);

		hipMemcpy(dev_a, a, size,hipMemcpyHostToDevice);
		hipMemcpy(dev_b, b, size,hipMemcpyHostToDevice);

		vecAdd<<<(int)ceil(N/T),T>>>(dev_a,dev_b,dev_c);

		hipMemcpy(c, dev_c, size,hipMemcpyDeviceToHost);

		hipFree(dev_a);
		hipFree(dev_b);
	    hipFree(dev_c);
	    t_exec=clock()-t_exec;
	    printArray(a,b,c);
	    printf("%d %f\n",T, (double)t_exec/CLOCKS_PER_SEC);
	}

	exit (0);
}

__global__ void vecAdd (int *a, int *b, int *c) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < N) {
		c[i] = a[i] + b[i];
	}
}

void printArray(int a[], int b[], int c[]) {

	// printf("a + b = c:\n");
	for(int i = 0; i < N; i++){
		//printf("%d + %d = %d\n", a[i],b[i],c[i]);
    }
	//printf("\n");
}
