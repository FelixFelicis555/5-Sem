#include "hip/hip_runtime.h"
#define N 2048

#include <stdio.h>
__global__ void matrixAdd (int *a, int *b, int *c);

int main() {
	int a[N][N], b[N][N], c[N][N];
	int *dev_a, *dev_b, *dev_c;
	int size = N * N * sizeof(int);

	// initialize a and b with real values (NOT SHOWN)
	for (int i = 0; i < N; i++) {
		for (int j = 0; j < N; j++) {
			a[i][j] = i+j;
			b[i][j] = i/2 + j/2;
			c[i][j] = 0;
		}
	}
	for (int k = 2; k <= 40; k+=1) {
		clock_t t_exec = clock();

		hipMalloc((void**)&dev_a, size);
		hipMalloc((void**)&dev_b, size);
		hipMalloc((void**)&dev_c, size);

		hipMemcpy(dev_a, a, size, hipMemcpyHostToDevice);
		hipMemcpy(dev_b, b, size, hipMemcpyHostToDevice);

		dim3 dimBlock(k, k);
		dim3 dimGrid((int)ceil(N/dimBlock.x),(int)ceil(N/dimBlock.y));

		matrixAdd<<<dimGrid,dimBlock>>>(dev_a,dev_b,dev_c);

		hipMemcpy(c, dev_c, size, hipMemcpyDeviceToHost);

		for (int i = 0; i < N; i++) {
			for (int j = 0; j < N; j++) {
				// printf("%d + %d = %d\n", a[i][j], b[i][j], c[i][j]);
			}
		}

		hipFree(dev_a);
		hipFree(dev_b);
		hipFree(dev_c);
		printf("%d %f\n", k, double(clock() - t_exec) / CLOCKS_PER_SEC);
	}
}

__global__ void matrixAdd (int *a, int *b, int *c) {
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int index = col + row * N;
	if (col < N && row < N) {
		c[index] = a[index] + b[index];
		// printf("%d", c[index]);
	}
}
