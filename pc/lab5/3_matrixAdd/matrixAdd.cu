#define N 512
#define BLOCK_DIM 512

#include <hip/hip_runtime.h>
#include <stdio.h>
__global__ void matrixAdd (int *a, int *b, int *c);

int main() {
	int a[N][N], b[N][N], c[N][N];
	int *dev_a, *dev_b, *dev_c;
	int size = N * N * sizeof(int);

	// initialize a and b with real values (NOT SHOWN)
	for (int i = 0; i < 200; i++) {
		for (int j = 0; j < 200; j++) {
			a[i][j] = i+j;
			b[i][j] = i/2 + j/2;
		}
	}

	hipMalloc((void**)&dev_a, size);
	hipMalloc((void**)&dev_b, size);
	hipMalloc((void**)&dev_c, size);

	hipMemcpy(dev_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, size, hipMemcpyHostToDevice);

	dim3 dimBlock(BLOCK_DIM, BLOCK_DIM);
	dim3 dimGrid((int)ceil(N/dimBlock.x),(int)ceil(N/dimBlock.y));

	matrixAdd<<<dimGrid,dimBlock>>>(dev_a,dev_b,dev_c);

	hipMemcpy(c, dev_c, size, hipMemcpyDeviceToHost);

	for (int i = 0; i < 200; i++) {
		for (int j = 0; j < 200; j++) {
			printf("%d + %d = %d\n", a[i][j], b[i][j], c[i][j]);
		}
	}

	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
}

__global__ void matrixAdd (int *a, int *b, int *c) {
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int index = col + row * N;
	if (col < N && row < N) {
		c[index] = a[index] + b[index];
		printf("%d", c[index]);
	}
}
