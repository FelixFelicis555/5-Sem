#define N 32
#define BLOCK_DIM 32

#include <hip/hip_runtime.h>
#include <stdio.h>
__global__ void matrixAdd (int *a, int *b, int *c);

int main() {
	int a[N][N], b[N][N], c[N][N];
	int *dev_a, *dev_b, *dev_c;
	int size = N * N * sizeof(int);

	// initialize a and b with real values (NOT SHOWN)
	for (int i = 0; i < N; i++) {
		for (int j = 0; j < N; j++) {
			a[i][j] = i+j;
			b[i][j] = i/2 + j/2;
			c[i][j] = 0;
		}
	}

	hipMalloc((void**)&dev_a, size);
	hipMalloc((void**)&dev_b, size);
	hipMalloc((void**)&dev_c, size);

	hipMemcpy(dev_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, size, hipMemcpyHostToDevice);

	dim3 dimBlock(BLOCK_DIM, BLOCK_DIM);
	dim3 dimGrid((int)ceil(N/dimBlock.x),(int)ceil(N/dimBlock.y));

	matrixAdd<<<dimGrid,dimBlock>>>(dev_a,dev_b,dev_c);

	hipMemcpy(c, dev_c, size, hipMemcpyDeviceToHost);

	for (int i = 0; i < N; i++) {
		for (int j = 0; j < N; j++) {
			printf("%d + %d = %d\n", a[i][j], b[i][j], c[i][j]);
		}
	}

	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
}

__global__ void matrixAdd (int *a, int *b, int *c) {
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int index = col + row * N;
	if (col < N && row < N) {
		c[index] = a[index] + b[index];
		// printf("%d", c[index]);
	}
}
