#include "hip/hip_runtime.h"
#define N 2048
#include <stdio.h>

__global__ void matrixMult (int *a, int *b, int *c, int width);

int main() {
	int a[N][N], b[N][N], c[N][N];
	int *dev_a, *dev_b, *dev_c;

	// initialize matrices a and b with appropriate values
	for (int i = 0; i < N; i++) {
		for (int j = 0; j < N; j++) {
			a[i][j] = i+j;
			b[i][j] = i-j;
			c[i][j] = 0;
		}
	}

	int size = N * N * sizeof(int);
	for(int BLOCK_DIM=2;BLOCK_DIM<=64;BLOCK_DIM+=2){
				clock_t t_exec = clock();

	hipMalloc((void **) &dev_a, size);
	hipMalloc((void **) &dev_b, size);
	hipMalloc((void **) &dev_c, size);

	hipMemcpy(dev_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, size, hipMemcpyHostToDevice);

	dim3 dimBlock(BLOCK_DIM, BLOCK_DIM);
	dim3 dimGrid((int)ceil(N/dimBlock.x),(int)ceil(N/dimBlock.y));

	matrixMult<<<dimGrid, dimBlock>>>(dev_a, dev_b, dev_c, N);

	hipMemcpy(c, dev_c, size, hipMemcpyDeviceToHost);


	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
	printf("%d %f\n", BLOCK_DIM, double(clock() - t_exec) / CLOCKS_PER_SEC);

}
	exit(0);
}

__global__ void matrixMult (int *a, int *b, int *c, int width) {
	int k, sum = 0;
	int col = threadIdx.x + blockDim.x * blockIdx.x;
	int row = threadIdx.y + blockDim.y * blockIdx.y;

	if(col < width && row < width) {
		for (k = 0; k < width; k++) {
			sum += a[row * width + k] * b[k * width + col];
		}
		c[row * width + col] = sum;
	}
}
