#define N 8
#define BLOCK_DIM 8

#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void matrixMult (int *a, int *b, int *c, int width);

int main() {
	int a[N][N], b[N][N], c[N][N];
	int *dev_a, *dev_b, *dev_c;

	// initialize matrices a and b with appropriate values
	for (int i = 0; i < N; i++) {
		for (int j = 0; j < N; j++) {
			a[i][j] = i+j;
			b[i][j] = i-j;
			c[i][j] = 0;
		}
	}

	int size = N * N * sizeof(int);

	hipMalloc((void **) &dev_a, size);
	hipMalloc((void **) &dev_b, size);
	hipMalloc((void **) &dev_c, size);

	hipMemcpy(dev_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, size, hipMemcpyHostToDevice);

	dim3 dimBlock(BLOCK_DIM, BLOCK_DIM);
	dim3 dimGrid((int)ceil(N/dimBlock.x),(int)ceil(N/dimBlock.y));

	matrixMult<<<dimGrid, dimBlock>>>(dev_a, dev_b, dev_c, N);

	hipMemcpy(c, dev_c, size, hipMemcpyDeviceToHost);

	for (int j = 0; j < N; j++) {
			printf("____");
		}
		printf("__\n");
	for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++){
            printf("|%-3d", a[i][j]);
        }
		printf(" |\n");
		for (int j = 0; j < N; j++) {
			printf("|___");
		}
		printf("_|\n");
    }
	printf("\n");
	for (int j = 0; j < N; j++) {
			printf("____");
		}
		printf("__\n");
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++){
            printf("|%2d ", b[i][j]);
        }
		printf(" |\n");
		for (int j = 0; j < N; j++) {
			printf("|___");
		}
		printf("_|\n");
    }
	printf("\n");
	for (int j = 0; j < N; j++) {
			printf("_______");
		}
		printf("__\n");
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++){
            printf("|%5d ", c[i][j]);
        }
		printf(" |\n");
		for (int j = 0; j < N; j++) {
			printf("|______");
		}
		printf("_|\n");
    }

	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);

	exit(0);
}

__global__ void matrixMult (int *a, int *b, int *c, int width) {
	int k, sum = 0;
	int col = threadIdx.x + blockDim.x * blockIdx.x;
	int row = threadIdx.y + blockDim.y * blockIdx.y;

	if(col < width && row < width) {
		for (k = 0; k < width; k++) {
			sum += a[row * width + k] * b[k * width + col];
		}
		c[row * width + col] = sum;
	}
}
