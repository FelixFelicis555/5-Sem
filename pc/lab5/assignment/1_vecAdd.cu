#include "hip/hip_runtime.h"
#define N 256
#include <stdio.h>
__global__ void vecAdd (int *a, int *b, int *c);
void printArray(int a[N], int b[N], int c[N]);

int main() {
	int a[N], b[N], c[N];
	int *dev_a, *dev_b, *dev_c;
	// initialize a and b with real values (NOT SHOWN)
	int size = N * sizeof(int);
	for (int i = 0; i < 200; i++) {
		a[i] = i;
		b[i] = i/2;
	}
	hipMalloc((void**)&dev_a, size);
	hipMalloc((void**)&dev_b, size);
	hipMalloc((void**)&dev_c, size);

	hipMemcpy(dev_a, a, size,hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, size,hipMemcpyHostToDevice);
	vecAdd<<<1,N>>>(dev_a,dev_b,dev_c);
	hipMemcpy(c, dev_c, size,hipMemcpyDeviceToHost);

	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);

	printArray(a,b,c);
	
	exit (0);
}

__global__ void vecAdd (int *a, int *b, int *c) {
	int i = threadIdx.x;
	c[i] = a[i] + b[i];
}

void printArray(int a[N], int b[N], int c[N]) {

	printf("Array a:\n")
	for(int i = 0; i < N; i++){
		printf("%d ", a[i]);
    }
    printf("\n\nArray b:\n");
    for(int i = 0; i < N; i++){
		printf("%d ", b[i]);
    }
    printf("\n\nArray c:\n");
    for(int i = 0; i < N; i++){
		printf("%d ", c[i]);
    }
}
