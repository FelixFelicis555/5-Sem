#include "hip/hip_runtime.h"
#include<stdio.h>
#define N 16
#define BLOCK_DIM 16

__global__ void matrixAdd (int *a, int *b, int *c);

int main() {
    int a[N][N], b[N][N], c[N][N];
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++){
            a[i][j] = i+j;
            b[i][j] = 0;
            c[i][j] = 0;
        }
        b[i][i] = 2;
    }
    int *dev_a, *dev_b, *dev_c;
    int size = N * N * sizeof(int);

    // initialize a and b with real values (NOT SHOWN)
    hipMalloc((void**) &dev_a, size);
    hipMalloc((void**) &dev_b, size);
    hipMalloc((void**) &dev_c, size);

    hipMemcpy (dev_a, a, size, hipMemcpyHostToDevice) ;
    hipMemcpy (dev_b, b, size, hipMemcpyHostToDevice) ;
    dim3 dimBlock(BLOCK_DIM, BLOCK_DIM) ;

    dim3 dimGrid( (int) ceil (N/dimBlock.x) , (int) ceil (N/dimBlock.y));
    matrixAdd<<<dimGrid, dimBlock>>> (dev_a,dev_b,dev_c);
    hipMemcpy(c, dev_c, size, hipMemcpyDeviceToHost) ;
    hipFree(dev_a); hipFree(dev_b); hipFree (dev_c) ;

    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++){
            printf("%d ", a[i][j]);
        }
        printf("\n");
    }
    printf("\n");
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++){
            printf("%d ", b[i][j]);
        }
        printf("\n");
    }
    printf("\n");
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++){
            printf("%d ", c[i][j]);
        }
        printf("\n");
    }
}

__global__ void matrixAdd (int *a, int *b, int *e) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int index = col + row * N;
    if (col < N && row < N) {
        e[index] = a[index] + b[index] ;
        //printf("a[i][j] = %d, b[i][j] = %d, i,j = %d,%d\n", a[i][j], b[i][j], i, j);
    }
}

void printArray(int a[N], int b[N], int c[N]) {

	for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++){
            printf("%d ", a[i][j]);
        }
        printf("\n");
    }
    printf("\n");
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++){
            printf("%d ", b[i][j]);
        }
        printf("\n");
    }
    printf("\n");
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++){
            printf("%d ", c[i][j]);
        }
        printf("\n");
    }

}
