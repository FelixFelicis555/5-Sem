
#include <hip/hip_runtime.h>
#include<stdio.h>
#define N 16
#define BLOCK_DIM 16

__global__ void matrixmul (int *a, int *b, int *c,int width);

int main() {
    int a[N][N], b[N][N], c[N][N];
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++){
            a[i][j] = i+j;
            b[i][j] = 0;
            c[i][j] = 0;
        }
        b[i][i] = 2;
    }
    int *dev_a, *dev_b, *dev_c;
    int size = N * N * sizeof(int);

    // initialize a and b with real values (NOT SHOWN)
    hipMalloc((void**) &dev_a, size);
    hipMalloc((void**) &dev_b, size);
    hipMalloc((void**) &dev_c, size);

    hipMemcpy (dev_a, a, size, hipMemcpyHostToDevice) ;
    hipMemcpy (dev_b, b, size, hipMemcpyHostToDevice) ;
    dim3 dimGrid(1,1);
    dim3 dimBlock(BLOCK_DIM, BLOCK_DIM) ;

    matrixmul<<<dimGrid, dimBlock>>> (dev_a,dev_b,dev_c,N);
    hipMemcpy(c, dev_c, size, hipMemcpyDeviceToHost) ;
    hipFree(dev_a); hipFree(dev_b); hipFree (dev_c) ;

    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++){
            printf("%d ", a[i][j]);
        }
        printf("\n");
    }
    printf("\n");
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++){
            printf("%d ", b[i][j]);
        }
        printf("\n");
    }
    printf("\n");
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++){
            printf("%d ", c[i][j]);
        }
        printf("\n");
    }
}

__global__ void matrixmul (int *a, int *b, int *e,int width) {
    int k,sum=0;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    if (col < width && row < width) {
        for(int k=0;k<width;k++)
            sum+=a[row*width+k]*b[k*width+col];
        e[row*width+col] = sum;
    }
}