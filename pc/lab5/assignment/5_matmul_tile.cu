
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#define N 16
#define BLOCK_DIM 16


#define BLK_ROWS 2
#define BLK_COLS 2
//size of the share memory tile in the device
#define TILE_SIZE BLK_ROWS
__global__ void matrixmul (int *a, int *b, int *c,int a_rows, int a_columns, int b_columns);

int main() {
    int a[N][N], b[N][N], c[N][N];
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++){
            a[i][j] = i+j;
            b[i][j] = 0;
            c[i][j] = 0;
        }
        b[i][i] = 2;
    }
    int *dev_a, *dev_b, *dev_c;
    int size = N * N * sizeof(int);

    // initialize a and b with real values (NOT SHOWN)
    hipMalloc((void**) &dev_a, size);
    hipMalloc((void**) &dev_b, size);
    hipMalloc((void**) &dev_c, size);

    hipMemcpy (dev_a, a, size, hipMemcpyHostToDevice) ;
    hipMemcpy (dev_b, b, size, hipMemcpyHostToDevice) ;

    dim3 dimBlock(BLK_COLS,BLK_ROWS);
    dim3 dimGrid((int)ceil(N/BLK_COLS),(int)ceil(N/BLK_ROWS));

    matrixmul<<<dimGrid, dimBlock>>> (dev_a,dev_b,dev_c,N,N,N);
    hipMemcpy(c, dev_c, size, hipMemcpyDeviceToHost) ;
    hipFree(dev_a); hipFree(dev_b); hipFree (dev_c) ;

    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++){
            printf("%d ", a[i][j]);
        }
        printf("\n");
    }
    printf("\n");
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++){
            printf("%d ", b[i][j]);
        }
        printf("\n");
    }
    printf("\n");
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++){
            printf("%d ", c[i][j]);
        }
        printf("\n");
    }
}

__global__ void matrixmul (int *a, int *b, int *c,int a_rows, int a_columns, int b_columns) {
    //declare shared memory matrices for A and B matrices
	__shared__ int shared_a_tile[TILE_SIZE][TILE_SIZE];
	__shared__ int shared_b_tile[TILE_SIZE][TILE_SIZE];

	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int row = blockIdx.y * blockDim.y + threadIdx.y;

	//check if thread directly maps to the dimensions of the resulting matrix
	if (row < a_rows && col < b_columns)
	{
		int result = 0;
		int k;
		int phase;

		//calculate C matrix indexes in phases. Each phase shares
		//TILE_SIZE * TILE_SIZE data copied to the shared matrix A
		//and matrix B.
		for (phase = 0; phase <= a_columns/TILE_SIZE; phase++)
		{
			shared_a_tile[ty][tx] = a[row * a_columns + phase * TILE_SIZE + tx];
			shared_b_tile[ty][tx] = b[(phase * TILE_SIZE + ty) * b_columns + col];
			__syncthreads();

			for (k = 0; k < TILE_SIZE; k++)
			{
				if (k + (phase * TILE_SIZE) < a_columns)
				{
					result += (shared_a_tile[ty][k] * shared_b_tile[k][tx]);
				}
			}
			__syncthreads();
		}
		c[row * b_columns + col] = result;
	}
}
